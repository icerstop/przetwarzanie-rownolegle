#include "hip/hip_runtime.h"
// Jeden plik: SEQ + OpenMP + CUDA TSP na tych samych podzbiorach

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#ifdef _OPENMP
  #include <omp.h>
#endif
#include <hip/hip_runtime.h>

// --------------------------------------------------
// Struktury i funkcje wsp�lne
// --------------------------------------------------
typedef struct {
    double lat, lon;
    char name[64];  // Dodane pole dla nazwy miasta
} City;

static inline double to_rad(double deg) {
    return deg * M_PI / 180.0;
}

static inline double haversine(const City *a, const City *b) {
    double dlat = to_rad(b->lat - a->lat);
    double dlon = to_rad(b->lon - a->lon);
    double rlat1 = to_rad(a->lat), rlat2 = to_rad(b->lat);
    double sdlat = sin(dlat/2), sdlon = sin(dlon/2);
    double h = sdlat*sdlat + sdlon*sdlon * cos(rlat1)*cos(rlat2);
    return 2 * 6371.0 * asin(sqrt(h));
}

void load_cities(const char *fname, City **out, int *n) {
    FILE *f = fopen(fname, "r");
    if (!f) {
        fprintf(stderr, "Error: Cannot open file '%s'\n", fname);
        exit(1);
    }
    char line[256];
    // Pomijamy nag��wek
    fgets(line, sizeof(line), f);
    int count = 0;
    while (fgets(line, sizeof(line), f)){
        char *tok = strtok(line, ",");
        tok = strtok(NULL, ",");
        if (!tok) break;
        tok = strtok(NULL, ",");
        if (!tok) break;
        count++;
    }
    rewind(f);
    fgets(line, sizeof(line), f);

    City *arr = (City*)malloc(count * sizeof(City));
    if (!arr){
        fprintf(stderr, "Error: Cannot allocate memory for %d cities\n", count);
        fclose(f);
        exit(1);
    }
    int i = 0;
    while (i < count && fgets(line, sizeof(line), f)) {
        char *tok = strtok(line, ",");        // nazwa miasta
        if (tok) {
            strncpy(arr[i].name, tok, sizeof(arr[i].name) - 1);
            arr[i].name[sizeof(arr[i].name) - 1] = '\0';
        } else {
            snprintf(arr[i].name, sizeof(arr[i].name), "City_%d", i);
        }

        tok = strtok(NULL, ",");
        if (!tok) break;
        arr[i].lat = atof(tok);
        tok = strtok(NULL, ",");
        if (!tok) break;
        arr[i].lon = atof(tok);
        i++;
    }
    fclose(f);
    *out = arr;
    *n = i;
}

void compute_dist(const City *cities, double *dist, int N) {
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            dist[i*N + j] = haversine(&cities[i], &cities[j]);
}

// --------------------------------------------------
// Funkcja do obliczania dystansu trasy
// --------------------------------------------------
double calculate_tour_distance(const int *tour, const double *dist, int N) {
    double total_distance = 0.0;
    for (int i = 0; i < N; i++) {
        int from = tour[i];
        int to = (i + 1 < N) ? tour[i + 1] : tour[0];  // Powr�t do pocz�tku
        total_distance += dist[from * N + to];
    }
    return total_distance;
}

// --------------------------------------------------
// Funkcja do wy�wietlania trasy
// --------------------------------------------------
void print_route(const int *tour, const City *cities, const double *dist, int N, const char *algorithm) {
    double distance = calculate_tour_distance(tour, dist, N);

    if (N <= 25) {  // Szczeg�owa trasa tylko dla ma�ych instancji
        printf("%s Route: ", algorithm);
        for (int i = 0; i < N; i++) {
            printf("%s", cities[tour[i]].name);
            if (i < N - 1) printf(" -> ");
        }
        printf(" -> %s (%.2f km)\n", cities[tour[0]].name, distance);
    } else {  // Tylko dystans dla wi�kszych instancji
        printf("%s Route distance: %.2f km\n", algorithm, distance);
    }
    fflush(stdout);
}

// --------------------------------------------------
// SEQ: Nearest Neighbor + 2-opt
// --------------------------------------------------
void nearest_neighbor(int *tour, const double *dist, int N) {
    int *used = (int*)calloc(N, sizeof(int));
    tour[0] = 0; used[0] = 1;
    for (int i = 1; i < N; i++) {
        int prev = tour[i-1], best = -1;
        double bestd = 1e9;
        for (int j = 0; j < N; j++) {
            if (!used[j] && dist[prev*N + j] < bestd) {
                bestd = dist[prev*N + j];
                best = j;
            }
        }
        tour[i] = best;
        used[best] = 1;
    }
    free(used);
}

void two_opt_seq(int *tour, const double *dist, int N) {
    int improved = 1;
    while (improved) {
        improved = 0;
        for (int i = 1; i < N - 1 && !improved; i++) {
            for (int j = i + 1; j < N; j++) {
                int a = tour[i-1], b = tour[i],
                    c = tour[j],   d = (j+1<N ? tour[j+1] : tour[0]);
                double delta = dist[a*N + c]
                             + dist[b*N + d]
                             - dist[a*N + b]
                             - dist[c*N + d];
                if (delta < -1e-9) {
                    for (int x = i, y = j; x < y; x++, y--) {
                        int tmp = tour[x]; tour[x] = tour[y]; tour[y] = tmp;
                    }
                    improved = 1;
                    break;
                }
            }
        }
    }
}

// --------------------------------------------------
// OpenMP: r�wnoleg�e szukanie najlepszego swapu
// --------------------------------------------------
void two_opt_omp(int *tour, const double *dist, int N) {
    int improved = 1;

    while (improved) {
        improved = 0;
        int best_i = 0, best_j = 0;
        double best_delta = 0.0;

        // r�wnoleg�a sekcja do wyszukania najlepszego lokalnego ulepszenia
        #pragma omp parallel default(none) shared(tour, dist, N, best_delta, best_i, best_j)
        {
            int loc_i = 0, loc_j = 0;
            double loc_best = 0.0;

            // r�wnoleg�e roz�o�enie iteracji z pomini�ciem synchronizacji ko�ca p�tli
            #pragma omp for nowait
            for (int i = 1; i < N - 1; i++) {
                for (int j = i + 1; j < N; j++) {
                    int a = tour[i - 1], b = tour[i];
                    int c = tour[j], d = (j + 1 < N ? tour[j + 1] : tour[0]);

                    double delta = dist[a * N + c] + dist[b * N + d]
                                 - dist[a * N + b] - dist[c * N + d];

                    if (delta < loc_best) {
                        loc_best = delta;
                        loc_i = i;
                        loc_j = j;
                    }
                }
            }

            // sekcja krytyczna � aktualizacja najlepszej znanej poprawy
            #pragma omp critical
            {
                if (loc_best < best_delta) {
                    best_delta = loc_best;
                    best_i = loc_i;
                    best_j = loc_j;
                }
            }
        }

        // Zastosowanie najlepszego znalezionego ulepszenia (je�li wyst�puje)
        if (best_delta < -1e-9) {
            for (int x = best_i, y = best_j; x < y; x++, y--) {
                int tmp = tour[x];
                tour[x] = tour[y];
                tour[y] = tmp;
            }
            improved = 1;
        }
    }
}
// --------------------------------------------------
// CUDA: atomowa operacja min dla double
// --------------------------------------------------
__device__ double atomicMinDouble(double *addr, double val) {
    unsigned long long *ptr = (unsigned long long*)addr;
    unsigned long long old = *ptr, assumed;
    do {
        assumed = old;
        if (__longlong_as_double(assumed) <= val) break;
        old = atomicCAS(ptr, assumed, __double_as_longlong(val));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// --------------------------------------------------
// CUDA kernel: wyszukaj najlepszy swap
// --------------------------------------------------
__global__ void find_best_swap(int *tour, double *dist, int N,
                               int *best_i, int *best_j, double *best_delta) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = (N-1)*N/2;
    if (idx >= total) return;

    // zamapuj idx -> (i,j)
    int i = 1, rem = idx;
    while (rem >= N - i) { rem -= (N - i); i++; }
    int j = i + 1 + rem;

    int a = tour[i-1], b = tour[i],
        c = tour[j],   d = (j+1<N ? tour[j+1] : tour[0]);
    double delta = dist[a*N + c]
                 + dist[b*N + d]
                 - dist[a*N + b]
                 - dist[c*N + d];

    // atomowo zaktualizuj najlepsz� popraw�
    double prev = atomicMinDouble(best_delta, delta);
    if (prev > delta) {
        atomicExch(best_i, i);
        atomicExch(best_j, j);
    }
}

// --------------------------------------------------
// GPU: 2-opt z kernelem
// --------------------------------------------------
void two_opt_cuda(int *tour, double *dist, int N) {
    int *d_tour, *d_best_i, *d_best_j;
    double *d_dist, *d_best_delta;
    int total_pairs = (N-1)*N/2;

    hipMalloc(&d_tour,       N * sizeof(int));
    hipMalloc(&d_dist,       N * N * sizeof(double));
    hipMalloc(&d_best_i,     sizeof(int));
    hipMalloc(&d_best_j,     sizeof(int));
    hipMalloc(&d_best_delta, sizeof(double));

    hipMemcpy(d_tour, tour, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dist, dist, N * N * sizeof(double), hipMemcpyHostToDevice);

    int improved = 1;
    while (improved) {
        improved = 0;
        double h_delta = 0.0;
        int h_i = 0, h_j = 0;
        hipMemcpy(d_best_delta, &h_delta, sizeof(double), hipMemcpyHostToDevice);

        int threads = 1024;
        int blocks  = (total_pairs + threads - 1) / threads;
        find_best_swap<<<blocks, threads>>>(d_tour, d_dist, N,
                                            d_best_i, d_best_j, d_best_delta);
        hipDeviceSynchronize();

        hipMemcpy(&h_delta,   d_best_delta, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&h_i,       d_best_i,     sizeof(int),    hipMemcpyDeviceToHost);
        hipMemcpy(&h_j,       d_best_j,     sizeof(int),    hipMemcpyDeviceToHost);

        if (h_delta < -1e-9) {
            // zastosuj swap na CPU
            for (int x = h_i, y = h_j; x < y; x++, y--) {
                int tmp = tour[x]; tour[x] = tour[y]; tour[y] = tmp;
            }
            // zsynchronizuj z GPU i powt�rz
            hipMemcpy(d_tour, tour, N * sizeof(int), hipMemcpyHostToDevice);
            improved = 1;
        }
    }

    hipMemcpy(tour, d_tour, N * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_tour);       hipFree(d_dist);
    hipFree(d_best_i);     hipFree(d_best_j);
    hipFree(d_best_delta);
}

// --------------------------------------------------
// main(): argumenty z linii komend
// --------------------------------------------------
int main(int argc, char *argv[]) {
    if (argc < 3) {
        fprintf(stderr, "Usage: %s <cities_file> <size1> <size2> ... <sizeN>\n", argv[0]);
        fprintf(stderr, "Example: %s cities.csv 10 25 50 150 250\n", argv[0]);
        return 1;
    }

    const char *filename = argv[1];
    int num_sizes = argc - 2;
    int *sizes = (int*)malloc(num_sizes * sizeof(int));

    // Parsowanie rozmiar�w z argument�w
    for (int i = 0; i < num_sizes; i++) {
        sizes[i] = atoi(argv[i + 2]);
        if (sizes[i] <= 0) {
            fprintf(stderr, "Error: Size must be positive integer, got '%s'\n", argv[i + 2]);
            free(sizes);
            return 1;
        }
    }

    City *cities;
    int fullN;
    load_cities(filename, &cities, &fullN);
    printf("Loaded %d cities from '%s'\n", fullN, filename);

    // Sprawdzenie czy wszystkie rozmiary s� mo�liwe
    for (int i = 0; i < num_sizes; i++) {
        if (sizes[i] > fullN) {
            fprintf(stderr, "Error: Requested size %d is larger than available cities (%d)\n",
                    sizes[i], fullN);
            free(sizes);
            free(cities);
            return 1;
        }
    }

    hipFree(0);
    int max_procs = omp_get_num_procs();
    omp_set_num_threads(max_procs);

    #pragma omp parallel
    {}

    {
        int dummyTour[2] = {0,1};
        double dummyDist[4] = {0.0,0.0,0.0,0.0};
        two_opt_omp(dummyTour, dummyDist, 2);
    }

    const int runs = 5;
    FILE *f;

    // nag��wek
    f = fopen("results.csv", "w");
    fprintf(f, "alg,size,time\n");
    fclose(f);

    srand(time(NULL));
    printf("Starting benchmark with %d runs for each size and algorithm\n", runs);

    for (int r = 0; r < runs; r++) {
        printf("Run %d/%d\n", r + 1, runs);
        for (int si = 0; si < num_sizes; si++) {
            int subN = sizes[si];

            // wsp�lny podzbi�r
            int *idx    = (int*)malloc(subN * sizeof(int));
            int *picked = (int*)calloc(fullN, sizeof(int));
            for (int i = 0; i < subN; i++) {
                int x;
                do { x = rand() % fullN; } while (picked[x]);
                picked[x] = 1;
                idx[i]    = x;
            }
            free(picked);

            // budujemy macierz odleg�o�ci
            City   *sub  = (City*)  malloc(subN * sizeof(City));
            double *dist = (double*)malloc(subN * subN * sizeof(double));
            for (int i = 0; i < subN; i++) sub[i] = cities[idx[i]];
            compute_dist(sub, dist, subN);

            int *tour = (int*)malloc(subN * sizeof(int));
            struct timespec t1, t2;
            double dt;

            // SEQ
            clock_gettime(CLOCK_MONOTONIC, &t1);
            nearest_neighbor(tour, dist, subN);
            two_opt_seq(tour, dist, subN);
            clock_gettime(CLOCK_MONOTONIC, &t2);
            dt = (t2.tv_sec - t1.tv_sec) + (t2.tv_nsec - t1.tv_nsec)/1e9;
            f = fopen("results.csv","a");
            fprintf(f, "SEQ,%d,%.6f\n", subN, dt);
            fclose(f);
            printf("Done SEQ: %d cities, %.6f s\n", subN, dt);
            print_route(tour, sub, dist, subN, "SEQ");  // Wy�wietl tras� dla ma�ych instancji
            fflush(stdout);

            // OpenMP
            clock_gettime(CLOCK_MONOTONIC, &t1);
            nearest_neighbor(tour, dist, subN);
            two_opt_omp(tour, dist, subN);
            clock_gettime(CLOCK_MONOTONIC, &t2);
            dt = (t2.tv_sec - t1.tv_sec) + (t2.tv_nsec - t1.tv_nsec)/1e9;
            f = fopen("results.csv","a");
            fprintf(f, "OMP,%d,%.6f\n", subN, dt);
            fclose(f);
            printf("Done OMP: %d cities, %.6f s\n", subN, dt);
            print_route(tour, sub, dist, subN, "OMP");  // Wy�wietl tras� dla ma�ych instancji
            fflush(stdout);

            // CUDA
            clock_gettime(CLOCK_MONOTONIC, &t1);
            nearest_neighbor(tour, dist, subN);
            two_opt_cuda(tour, dist, subN);
            clock_gettime(CLOCK_MONOTONIC, &t2);
            dt = (t2.tv_sec - t1.tv_sec) + (t2.tv_nsec - t1.tv_nsec)/1e9;
            f = fopen("results.csv","a");
            fprintf(f, "CUDA,%d,%.6f\n", subN, dt);
            fclose(f);
            printf("Done CUDA: %d cities, %.6f s\n", subN, dt);
            print_route(tour, sub, dist, subN, "CUDA");  // Wy�wietl tras� dla ma�ych instancji
            fflush(stdout);

            free(idx); free(sub); free(dist); free(tour);
        }
    }

    free(sizes);
    free(cities);
    return 0;
